// cuda_utils.cu

#include "../../include/gpu/cuda_utils.hpp"

#include <iostream>
#include <stdexcept>

// Definitions of utility functions

void initializeCUDAErrorHandling() {
    // Currently, CUDA does not support setting a custom error handler directly.
    // You can use hipDeviceSetLimit or other mechanisms if needed.
    // For now, we'll leave this function empty.
}

void printDeviceProperties() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t props;
        CUDA_CHECK(hipGetDeviceProperties(&props, device));

        std::cout << "\nCUDA Device " << device << ": " << props.name << std::endl;
        std::cout << "  Compute Capability: " << props.major << "." << props.minor << std::endl;
        std::cout << "  Total Global Memory: "
                  << props.totalGlobalMem / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "  Max Threads per Block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Threads per Multiprocessor: "
                  << props.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Number of Multiprocessors: " << props.multiProcessorCount << std::endl;
        std::cout << "  Warp Size: " << props.warpSize << std::endl;
        std::cout << "  Memory Clock Rate: "
                  << props.memoryClockRate / 1000.0 << " MHz" << std::endl;
        std::cout << "  Memory Bus Width: " << props.memoryBusWidth << " bits" << std::endl;
        std::cout << "  Peak Memory Bandwidth: "
                  << 2.0 * props.memoryClockRate * (props.memoryBusWidth / 8) / 1.0e6
                  << " GB/s" << std::endl;
    }
}

void* allocateDeviceMemory(size_t size) {
    void* ptr = nullptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

void freeDeviceMemory(void* ptr) {
    if (ptr) {
        CUDA_CHECK(hipFree(ptr));
    }
}

hipStream_t createCUDAStream() {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    return stream;
}

void destroyCUDAStream(hipStream_t stream) {
    if (stream) {
        CUDA_CHECK(hipStreamDestroy(stream));
    }
}

void synchronizeDevice() {
    CUDA_CHECK(hipDeviceSynchronize());
}

void copyToDevice(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void copyToHost(void* dst, const void* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

hipEvent_t createCUDAEvent() {
    hipEvent_t event;
    CUDA_CHECK(hipEventCreate(&event));
    return event;
}

void destroyCUDAEvent(hipEvent_t event) {
    if (event) {
        CUDA_CHECK(hipEventDestroy(event));
    }
}

float getEventElapsedTime(hipEvent_t start, hipEvent_t end) {
    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
    return ms;
}

// Optionally, you can include any other non-inline CUDA utility functions here.
// For example, device synchronization, error handling, etc.

